// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}




extern "C"{
    float* allocArray(int ln, float* values) {
        float* result = (float*) malloc(ln * sizeof(float));
        for (int i = 0; i < ln; i++) {
                result[i] = values[i];
        }
        return result;
    }
    void freeArr(float* p) { free(p); }
}


 extern "C" {
    typedef struct {//typedef
        float* numbers;
    	int numOfColumns;
    	int numOfRows;
    } Matrix;
}

void print_matrix_struct(Matrix *matrix) {

 for(int i = 0; i < (matrix->numOfColumns * matrix->numOfRows); ++i){
    std::cout << matrix->numbers[i] << " ";

 }

 std::cout << std::endl;


}

 extern "C" {
    void matrices(void) {
        // Allocate 3 arrays on CPU
        int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

        // for simplicity we are going to use square arrays
        nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;

        float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
        float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
        float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

        // Allocate 3 arrays on GPU
        float *d_A, *d_B, *d_C;
        hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
        hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
        hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

        // If you already have useful values in A and B you can copy them in GPU:
        // cudaMemcpy(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),cudaMemcpyHostToDevice);
        // cudaMemcpy(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),cudaMemcpyHostToDevice);

        // Fill the arrays A and B on GPU with random numbers
        GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
        GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

        // Optionally we can copy the data back on CPU and print the arrays
        hipMemcpy(h_A,d_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_B,d_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyDeviceToHost);
        std::cout << "A =" << std::endl;
        print_matrix(h_A, nr_rows_A, nr_cols_A);
        std::cout << "B =" << std::endl;
        print_matrix(h_B, nr_rows_B, nr_cols_B);

        // Multiply A and B on GPU
        gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

        // Copy (and print) the result on host memory
        hipMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost);
        std::cout << "C =" << std::endl;
        print_matrix(h_C, nr_rows_C, nr_cols_C);

        //Free GPU memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        // Free CPU memory
        free(h_A);
        free(h_B);
        free(h_C);

        //return 0;
    }

    void multiplyMatrices(Matrix *matrix1, Matrix *matrix2) {
            print_matrix_struct(matrix1);
            print_matrix_struct(matrix2);


            // Allocate 3 arrays on CPU
//            int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

            // for simplicity we are going to use square arrays
//            nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;

            /*float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
            float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));*/
            float *h_A = matrix1->numbers;
            float *h_B = matrix2->numbers;
            float *h_C = (float *)malloc(matrix1->numOfRows * matrix2->numOfColumns * sizeof(float));

            // Allocate 3 arrays on GPU
            float *d_A, *d_B, *d_C; //*d_A, *d_B,
            hipMalloc(&d_A,matrix1->numOfRows * matrix1->numOfColumns * sizeof(float));
            hipMalloc(&d_B,matrix2->numOfRows * matrix2->numOfColumns * sizeof(float));
            hipMalloc(&d_C,matrix1->numOfRows * matrix2->numOfColumns * sizeof(float));

            /**h_A = matrix1->numbers;
            *h_B = matrix2->numbers;*/



            // If you already have useful values in A and B you can copy them in GPU:
            hipMemcpy(d_A,h_A,matrix1->numOfRows * matrix1->numOfColumns * sizeof(float),hipMemcpyHostToDevice);
            hipMemcpy(d_B,h_B,matrix2->numOfRows * matrix2->numOfColumns * sizeof(float),hipMemcpyHostToDevice);

            // Fill the arrays A and B on GPU with random numbers
            /*GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
            GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);*/

            // Optionally we can copy the data back on CPU and print the arrays
            hipMemcpy(h_A,d_A,matrix1->numOfRows * matrix1->numOfColumns * sizeof(float),hipMemcpyDeviceToHost);
            hipMemcpy(h_B,d_B,matrix2->numOfRows * matrix2->numOfColumns * sizeof(float),hipMemcpyDeviceToHost);
            std::cout << "A =" << std::endl;
            print_matrix(h_A, matrix1->numOfRows, matrix1->numOfColumns);
            std::cout << "B =" << std::endl;
            print_matrix(h_B, matrix2->numOfRows, matrix2->numOfColumns);

            // Multiply A and B on GPU
//            gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
            gpu_blas_mmul(d_A, d_B, d_C, matrix1->numOfRows, matrix1->numOfColumns, matrix2->numOfColumns);

            // Copy (and print) the result on host memory
            hipMemcpy(h_C,d_C,matrix1->numOfRows * matrix2->numOfColumns * sizeof(float),hipMemcpyDeviceToHost);
            std::cout << "C =" << std::endl;
            print_matrix(h_C, matrix1->numOfRows, matrix2->numOfColumns);

            //Free GPU memory
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);

            // Free CPU memory
            free(h_A);
            free(h_B);
            free(h_C);

            //return 0;
        }
}