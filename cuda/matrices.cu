#include "hip/hip_runtime.h"
// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <vector>

char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}


struct sigmoid_functor
{
  __host__ __device__
  float operator()(const float& x) const
  {
       return 1/(1.0 + expf(-x));
  }
};

struct sigmoid_derivative_functor
{
  __host__ __device__
  float operator()(const float& x) const
  {
       return expf(x)/powf((1.0 + expf(x)),2);
  }
};




// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}

void printArray(float *array ,int length){
     std::cout << std::endl;
      std::cout << "array: ";
    for(int i = 0; i < length; ++i){
        std::cout << array[i] << " ";
    }
}

//y = α op(A)x + βy
void gpu_blas_matrix_with_vector(const float *A, const float *v, float *result, const int m, const int n) {

	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;
//	char* resultStatus;


	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);


	// Do the actual multiplication
	// matrix - vector multiplication : d_y = al*d_a *d_x + bet *d_y
    // d_a - mxn matrix ; d_x - n-vector , d_y - m- vector ;
    // al ,bet - scalars
    //hipblasSgemv(handle,CUBLAS OP N,m,n,&al,d a,m,d x,1,&bet,d y,1);
    /*resultStatus = cublasGetErrorString(hipblasSgemv(handle,HIPBLAS_OP_N,m,n,alpha,A,m,v,1,beta,result,1));
     std::cout << "CUblas result: \n" << resultStatus;
*/
    hipblasSgemv(handle,HIPBLAS_OP_N,m,n,alpha,A,m,v,1,beta,result,1);

	// Destroy the handle
	hipblasDestroy(handle);
}


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}




extern "C"{
    float* allocArray(int ln, float* values) {
        float* result = (float*) malloc(ln * sizeof(float));
        for (int i = 0; i < ln; i++) {
                result[i] = values[i];
        }
        return result;
    }

    void freeArr(float* p) { free(p); }

    float* allocEmptyArray(int ln) {
            float* result = (float*) malloc(ln * sizeof(float));

            return result;
    }

    void getNumbers(float *filled, float* empty, int numOfRows, int numOfColumns){
         for (int i = 0; i < (numOfRows * numOfColumns); i++) {
                        empty[i] = filled[i];
          }
    }
}


 extern "C" {
    typedef struct {//typedef
        float* numbers;
    	int numOfColumns;
    	int numOfRows;
    } Matrix;
}

void print_matrix_struct(Matrix *matrix) {
     for(int i = 0; i < (matrix->numOfColumns * matrix->numOfRows); ++i){
        std::cout << matrix->numbers[i] << " ";

     }
     std::cout << std::endl;
}

 extern "C" {

        void multiplyMatrices(Matrix *matrix1, Matrix *matrix2, Matrix *resultMatrix) {

                    thrust::device_vector<float> A(matrix1->numbers, matrix1->numbers+(matrix1->numOfRows * matrix1->numOfColumns));
                    thrust::device_vector<float> B(matrix2->numbers, matrix2->numbers+(matrix2->numOfRows * matrix2->numOfColumns));
                    thrust::device_vector<float> C(matrix1->numOfRows * matrix2->numOfColumns);


                    // Multiply A and B on GPU
                    gpu_blas_mmul(thrust::raw_pointer_cast(&A[0]), thrust::raw_pointer_cast(&B[0]), thrust::raw_pointer_cast(&C[0]), matrix1->numOfRows, matrix1->numOfColumns, matrix2->numOfColumns);


                    thrust::copy(C.begin(), C.end(), resultMatrix->numbers );

                    resultMatrix->numOfRows = matrix1->numOfRows;
                    resultMatrix->numOfColumns = matrix2->numOfColumns;

         }

         void multiplyMatrixWithVector(Matrix *matrix, float *vector, float *resultVector){

             thrust::device_vector<float> A(matrix->numbers, matrix->numbers+(matrix->numOfRows * matrix->numOfColumns));
             thrust::device_vector<float> v(vector, vector+matrix->numOfColumns);
             thrust::device_vector<float> result(matrix->numOfRows);

             gpu_blas_matrix_with_vector(thrust::raw_pointer_cast(&A[0]), thrust::raw_pointer_cast(&v[0]), thrust::raw_pointer_cast(&result[0]), matrix->numOfRows, matrix->numOfColumns);//matrix->numOfRows, matrix->numOfColumns

             thrust::copy(result.begin(), result.end(), resultVector);

         }

         void applySigmoidOnVector(float *vector,float *resultVector, int size){
             thrust::device_vector<float> v(vector, vector+size);
             thrust::device_vector<float> result(size);

             thrust::transform(v.begin(), v.end(), result.begin(), sigmoid_functor());

             thrust::copy(result.begin(), result.end(), resultVector);
         }

         void applySigmoidDerivativeOnVector(float *vector,float *resultVector, int size){
              thrust::device_vector<float> v(vector, vector+size);
              thrust::device_vector<float> result(size);

              thrust::transform(v.begin(), v.end(), result.begin(), sigmoid_derivative_functor());

              thrust::copy(result.begin(), result.end(), resultVector);
         }
}