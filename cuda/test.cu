#include "hip/hip_runtime.h"

#include <stdio.h>




__global__ void add(int *a, int *b, int *c){
     *c = *a + *b;
}


extern "C" {
    int test_addition(void) {

        printf("CUDA status: %d\n", hipDeviceSynchronize());

        int a, b, c; // host copies of a, b, c
            int *d_a, *d_b, *d_c; // device copies of a, b, c
            int size = sizeof(int);
            // Allocate space for device copies of a, b, c
            hipMalloc((void **)&d_a, size);
            hipMalloc((void **)&d_b, size);
            hipMalloc((void **)&d_c, size);
            // Setup input values
            a = 1;
            b = 7;

            // Copy inputs to device
            hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
            hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
            // Launch add() kernel on GPU
            add<<<1,1>>>(d_a, d_b, d_c);
            // Copy result back to host
            hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
            // Cleanup
            hipFree(d_a); hipFree(d_b); hipFree(d_c);

             printf("Result is: %d\n", c);


        return c;
     }
 }

 extern "C" {
     void count_devices(void) {
       int nDevices;

       hipGetDeviceCount(&nDevices);
       for (int i = 0; i < nDevices; i++) {
         hipDeviceProp_t prop;
         hipGetDeviceProperties(&prop, i);
         printf("Device Number: %d\n", i);
         printf("  Device name: %s\n", prop.name);
         printf("  Memory Clock Rate (KHz): %d\n",
                prop.memoryClockRate);
         printf("  Memory Bus Width (bits): %d\n",
                prop.memoryBusWidth);
         printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
                2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
       }
     }
 }